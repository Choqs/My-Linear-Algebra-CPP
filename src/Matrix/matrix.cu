#include "hip/hip_runtime.h"
template <class Number>
__host__ void addWrapper(Number* m1, Number* m2, Number* m3, size_t size) {

  // Pointer of arrays.
  Number* d_m1;
  Number* d_m2;
  Number* d_m3;

  // Allocating Device Memory.
  CudaMalloc(&d_m1, size * sizeof(Number));
  CudaMalloc(&d_m2, size * sizeof(Number));
  CudaMalloc(&d_m3, size * sizeof(Number));

  // Copying in Device Memory.
  hipMemcpy(d_m1, m1, hipMemcpyHostToDevice);
  hipMemcpy(d_m2, m2, hipMemcpyHostToDevice);
  hipMemcpy(d_m3, m3, hipMemcpyHostToDevice);

  // Calling the kernel function.
  addKernel<<<1, 1>>>(d_m1, d_m2, d_m3, size);

  // Copying in Host Memory.
  hipMemcpy(m3, d_m3, hipMemcpyDeviceToHost);

  // Freeing Device Memory.
  hipFree(d_m1);
  hipFree(d_m2);
  hipFree(d_m3);
}

template <class Number>
__global__ void addKernel(Number* m1, Number* m2, Number* m3, size_t size) {
  for (size_t i = 0; i < size; i++) {
    m3[i] = m1[i] + m2[i];
  }
}
