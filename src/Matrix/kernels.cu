#include <hip/hip_runtime.h>


#include <stdio.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

template <class Number>
__global__ void addKernel(Number* d_m1, Number* d_m2, Number* d_m3, size_t size) {
  for (size_t i = 0; i < size; i++) {
    d_m3[i] = d_m1[i] + d_m2[i];
  }
}

__host__ void addWrapper(int* m1, int* m2, int* m3, size_t size) {

  // Pointer of arrays.
  int* d_m1;
  int* d_m2;
  int* d_m3;

  // Allocating in Device Memory.
  hipMalloc(&d_m1, size * sizeof(int));
  hipMalloc(&d_m2, size * sizeof(int));
  hipMalloc(&d_m3, size * sizeof(int));

  // Copying in Device Memory.
  hipMemcpy(d_m1, m1, size * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_m2, m2, size * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_m3, m3, size * sizeof(int), hipMemcpyHostToDevice);

  // Calling the kernel function.
  addKernel<<<1, 1>>>(d_m1, d_m2, d_m3, size);
  hipDeviceSynchronize();

  // Copying Device Memory to Host Memory.
  hipMemcpy(m3, d_m3, size * sizeof(int), hipMemcpyDeviceToHost);

  // Freeing Device Memory.
  hipFree(d_m1);
  hipFree(d_m2);
  hipFree(d_m3);
}
