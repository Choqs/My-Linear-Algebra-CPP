#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>

#include "kernels.cuh"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

template <class Number>
__global__ void addKernel(Number* d_m1, Number* d_m2, Number* d_m3, size_t size) {
  for (size_t i = 0; i < size; i++) {
    d_m3[i] = d_m1[i] + d_m2[i];
  }
}

template <class Number>
__host__ void addKernelWrapper(Number* m1, Number* m2, Number* m3, size_t size) {
  // Pointer of arrays.
  int* d_m1;
  int* d_m2;
  int* d_m3;

  // Allocating in Device Memory.
  hipMalloc(&d_m1, size * sizeof(Number));
  hipMalloc(&d_m2, size * sizeof(Number));
  hipMalloc(&d_m3, size * sizeof(Number));

  // Copying in Device Memory.
  hipMemcpy(d_m1, m1, size * sizeof(Number), hipMemcpyHostToDevice);
  hipMemcpy(d_m2, m2, size * sizeof(Number), hipMemcpyHostToDevice);
  hipMemcpy(d_m3, m3, size * sizeof(Number), hipMemcpyHostToDevice);

  // Calling the kernel function.
  addKernel<<<1, 1>>>(d_m1, d_m2, d_m3, size);
  hipDeviceSynchronize();

  // Copying Device Memory to Host Memory.
  hipMemcpy(m3, d_m3, size * sizeof(Number), hipMemcpyDeviceToHost);

  // Freeing Device Memory.
  hipFree(d_m1);
  hipFree(d_m2);
  hipFree(d_m3);
}

template void Wrapper::add(int* m1, int* m2, int* m3, size_t size);
template void Wrapper::add(float* m1, float* m2, float* m3, size_t size);
template void Wrapper::add(double* m1, double* m2, double* m3, size_t size);
